#include "CubWrapper.cuh"
#include <hipcub/hipcub.hpp>


NAMESPACE_DPHPC_BEGIN

template <typename T> void DeviceSort(unsigned int numberOfElements, T** dKeysIn, T** dKeysOut,
                 unsigned int** dValuesIn, unsigned int** dValuesOut)
{   

    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<T> dKeys(*dKeysIn, *dKeysOut);
    hipcub::DoubleBuffer<unsigned int> dValues(*dValuesIn, *dValuesOut);
    
    // Determine temporary device storage requirements
    void     *dTempStorage = NULL;
    size_t   dTempStorageBytes = 0;
    hipcub::DeviceRadixSort::SortPairs(dTempStorage, dTempStorageBytes, dKeys, dValues, numberOfElements);
    // Allocate temporary storage
    hipMalloc(&dTempStorage, dTempStorageBytes);
    // Run sorting operation
    hipcub::DeviceRadixSort::SortPairs(dTempStorage, dTempStorageBytes, dKeys, dValues, numberOfElements);
    // Free temporary memory
    hipFree(dTempStorage);
    // Update out buffers
    T* current = dKeys.Current();
    dKeysOut = &current;
    unsigned int* current2 = dValues.Current();
    dValuesOut = &current2;

}

void DeviceSort(unsigned int numberOfElements, unsigned int** dKeysIn, unsigned int** dKeysOut,
                 unsigned int** dValuesIn, unsigned int** dValuesOut) {
    DeviceSort<unsigned int>(numberOfElements, dKeysIn, dKeysOut, dValuesIn, dValuesOut);
}

void DeviceSort(unsigned int numberOfElements, unsigned int* keysIn, unsigned int* keysOut,
    unsigned int* valuesIn, unsigned int* valuesOut)
{   

    // Allocate memory 
    unsigned int  *dKeysIn;
    unsigned int  *dKeysOut;
    unsigned int  *dValuesIn;
    unsigned int  *dValuesOut;

    hipMalloc(&dKeysIn, numberOfElements * sizeof(unsigned int));
    hipMalloc(&dKeysOut, numberOfElements * sizeof(unsigned int));
    hipMalloc(&dValuesIn, numberOfElements * sizeof(unsigned int));
    hipMalloc(&dValuesOut, numberOfElements * sizeof(unsigned int));

    // Copy input to Device
    hipMemcpy(dKeysIn, keysIn, sizeof(unsigned int) * numberOfElements, hipMemcpyHostToDevice);
    hipMemcpy(dValuesIn, valuesIn, sizeof(unsigned int) * numberOfElements, hipMemcpyHostToDevice);

    // Pereform sort on Device
    DeviceSort<unsigned int>(numberOfElements, &dKeysIn, &dKeysOut, &dValuesIn, &dValuesOut);

    // Copy results from Device to Host
    hipMemcpy(keysOut, dKeysOut, sizeof(unsigned int) * numberOfElements, hipMemcpyDeviceToHost);
    hipMemcpy(valuesOut, dValuesOut, sizeof(unsigned int) * numberOfElements, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(dKeysIn);
    hipFree(dKeysOut);
    hipFree(dValuesIn);
    hipFree(dValuesOut);

}

NAMESPACE_DPHPC_END
